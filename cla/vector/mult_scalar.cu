#include "hip/hip_runtime.h"
extern "C" {
#include "../include/cuda_utils.h"
#include "../include/entities.h"
#include "../include/vector_operations.h"
#include "../include/vector_utils.h"
}
__host__ __device__ void _vector_mult_scalar(double *a, Vector *b,
                                             Vector *dst) {
#if defined(__CUDA__ARCH__)
  return;
#else
  for (int i = 0; i < b->dims; i++) {
    dst->arr[i] = (*a) * b->arr[i];
  }
#endif
}

__global__ void _cu_vector_mult_scalar(double *a, Vector *b, Vector *dst) {
  _vector_mult_scalar(a, b, dst);
}

extern "C" Vector *vector_mult_scalar(double a, Vector *b, Vector *dst) {
  return cpu_gpu_conditional_apply_scalar_vector_operator(
      &_vector_mult_scalar, NULL, a, b, dst, b->dims, b->device);
}
