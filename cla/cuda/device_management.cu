extern "C" {
#include "../include/device_management.h"
#include "../include/entities.h"
#include "../include/vector_utils.h"
#include <assert.h>
#include <stdlib.h>
#include <string.h>
}

#include "hip/hip_runtime_api.h"

extern "C" void synchronize_devices() { hipDeviceSynchronize(); }

extern "C" void populate_devices() {
  if (DEVICES != NULL) {
    return;
  }

  // Initialize global variable
  DEVICES = (AvailableCUDADevices *)malloc(sizeof(AvailableCUDADevices));
  DEVICES->devices = NULL;
  DEVICES->count = 0;

  // Try to get CUDA devices count
  hipDeviceProp_t prop;
  hipGetDeviceCount(&DEVICES->count);

  // If it has devices, get their metadata
  if (DEVICES->count > 0) {
    DEVICES->devices =
        (CUDADevice *)malloc(sizeof(CUDADevice) * DEVICES->count);
    for (int i = 0; i < DEVICES->count; i++) {
      hipGetDeviceProperties(&prop, i);
      char *name = (char *)malloc(sizeof(char) * 256);
      strcpy(name, prop.name);
      DEVICES->devices[i] =
          (CUDADevice){i,
                       name,
                       prop.maxGridSize[0],
                       prop.maxGridSize[1],
                       prop.maxGridSize[2],
                       prop.maxThreadsDim[0],
                       prop.maxThreadsDim[1],
                       prop.maxThreadsDim[2],
                       prop.maxThreadsPerBlock,
                       (CUDAKernelLaunchParameters){0, 0, 0, 0, 0, 0}};
    }
  }
}

extern "C" void clear_devices() {
  // If no devices, return
  if (DEVICES == NULL) {
    return;
  }

  // If it has devices, deallocate memory
  //    for each one
  if (DEVICES->count > 0) {
    for (int i = 0; i < DEVICES->count; i++) {
      free(DEVICES->devices[i].name);
    }
    free(DEVICES->devices);
  }

  // Deallocate memory for global variable
  free(DEVICES);
  DEVICES = NULL;
}

extern "C" Vector *vector_to_cu(Vector *src, CUDADevice *device) {
  if (src->device != NULL) {
    // Maybe it's already on correct device?
    if (src->device == device) {
      return src;
    }

    // It's on another device!
    // Move data back to CPU before proceeding
    // Better handling requires CUDA Driver API
    vector_to_cpu(src);
  }

  // src is on CPU, activate target device
  hipSetDevice(device->id);

  // Allocate memory for a vector in GPU
  size_t vec_size = sizeof(Vector);
  Vector *cu_vector = NULL;
  hipMalloc(&cu_vector, vec_size);

  // Allocate memory for underlying array in GPU
  size_t arr_size = src->dims * sizeof(double);
  double *cu_arr = NULL;
  hipMalloc(&cu_arr, arr_size);

  // Copy array from CPU to GPU
  hipMemcpy(cu_arr, src->arr, arr_size, hipMemcpyHostToDevice);

  // Clear CPU array
  free(src->arr);

  // Copy vector from CPU to GPU in-place
  src->arr = cu_arr;
  src->cu_vector = NULL;
  src->device = NULL;
  hipMemcpy(cu_vector, src, vec_size, hipMemcpyHostToDevice);

  // Update CPU vector metadata
  src->device = device;
  src->arr = NULL;
  src->cu_vector = cu_vector;

  // Return CPU vector metadata
  return src;
}

extern "C" Vector *vector_to_cpu(Vector *src) {
  // Maybe device is already on CPU
  if (src->device == NULL) {
    return src;
  }

  // Guarantee CUDA uses the correct device
  hipSetDevice(src->device->id);

  // Preconditions
  assert(src->arr == NULL);

  // Allocate memory for underlying array in CPU
  size_t arr_size = src->dims * sizeof(double);
  double *vec_arr = (double *)malloc(arr_size);

  // Copy Vector from GPU to CPU in-place
  // We need to store the pointer to cu_vector
  //    since it will be override by the contents
  //    of cu_vector.
  Vector *cu_vector = src->cu_vector;
  hipMemcpy(src, cu_vector, sizeof(Vector), hipMemcpyDeviceToHost);

  // Copy the data from the cu_vector->arr to a CPU array
  double *cu_arr = src->arr;
  hipMemcpy(vec_arr, src->arr, arr_size, hipMemcpyDeviceToHost);

  // Clear GPU array
  hipFree(cu_arr);

  // Clear GPU vector
  hipFree(cu_vector);

  // Update CPU vector metadata
  src->arr = vec_arr;
  src->cu_vector = NULL;
  src->device = NULL;

  return src;
}

extern "C" Matrix *matrix_to_cu(Matrix *src, CUDADevice *device) {
  if (src->device != NULL) {
    // Maybe it's already on correct device?
    if (src->device == device) {
      return src;
    }

    // It's on another device!
    // Move data back to CPU before proceeding
    // Better handling requires CUDA Driver API
    matrix_to_cpu(src);
  }

  // src is on CPU, activate target device
  hipSetDevice(device->id);

  // 1. Allocate memory in GPU for the matrix
  size_t mat_size = sizeof(Matrix);
  Matrix *cu_matrix = NULL;
  hipMalloc(&cu_matrix, mat_size);

  // 2. Construct underlying array (i.e., a collection
  //    of n_rows pointers).
  size_t arr_size = src->rows * sizeof(double *);
  double **cu_arr = NULL;
  hipMalloc(&cu_arr, arr_size);

  // 3. Construct row arrays and copy them to
  //    cu_array.
  // tmp is needed to set each index of the
  //    the cu_array on CPU prior to copying
  //    it to GPU.
  size_t row_size = src->columns * sizeof(double);
  double **tmp = (double **)malloc(arr_size);
  for (int i = 0; i < src->rows; i++) {
    // Allocate memory on GPU for row
    double *cu_row = NULL;
    hipMalloc(&cu_row, row_size);

    // Copy the actual contents from src to
    //  the GPU row
    hipMemcpy(cu_row, src->arr[i], row_size, hipMemcpyHostToDevice);

    // Update the tmp with the address of this new
    //  row
    tmp[i] = cu_row;

    // Free the CPU memory
    free(src->arr[i]);
  }

  // Copy the tmp (array of GPU arrays addresses) to the
  //    GPU array
  hipMemcpy(cu_arr, tmp, arr_size, hipMemcpyHostToDevice);

  // Free CPU memory
  free(tmp);

  // 4. Now, we have the cu_array complete,
  //    all we need is to send a Matrix struct
  //    to GPU.
  free(src->arr);
  src->arr = cu_arr;
  src->cu_matrix = NULL;
  src->device = NULL;
  hipMemcpy(cu_matrix, src, mat_size, hipMemcpyHostToDevice);

  // Update CPU vector metadata
  src->device = device;
  src->arr = NULL;
  src->cu_matrix = cu_matrix;

  // Return CPU vector metadata
  return src;
}

extern "C" Matrix *matrix_to_cpu(Matrix *src) {
  // Maybe device is already on CPU
  if (src->device == NULL) {
    return src;
  }

  // Preconditions
  assert(src->arr == NULL);

  // Guarantee CUDA uses the correct device
  hipSetDevice(src->device->id);

  // 1. Bring back the Matrix from GPU
  size_t mat_size = sizeof(Matrix);
  Matrix *cu_matrix = src->cu_matrix;
  hipMemcpy(src, cu_matrix, mat_size, hipMemcpyDeviceToHost);
  hipFree(cu_matrix);

  // 2. Reconstruct array in CPU
  size_t arr_size = src->rows * sizeof(double *);
  size_t row_size = src->columns * sizeof(double);
  double **cu_arr = src->arr;
  src->arr = (double **)malloc(arr_size);
  hipMemcpy(src->arr, cu_arr, arr_size, hipMemcpyDeviceToHost);
  for (int i = 0; i < src->rows; i++) {
    double *row = (double *)malloc(row_size);
    double *cu_row = src->arr[i];
    hipMemcpy(row, cu_row, row_size, hipMemcpyDeviceToHost);
    src->arr[i] = row;
    hipFree(cu_row);
  }
  hipFree(cu_arr);

  // Update CPU vector metadata
  src->cu_matrix = NULL;
  src->device = NULL;

  return src;
}
