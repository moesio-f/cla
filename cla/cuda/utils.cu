#include "hip/hip_runtime.h"
extern "C" {
#include "../include/cuda_utils.h"
#include "../include/entities.h"
#include "../include/matrix_utils.h"
#include "../include/vector_utils.h"
#include <assert.h>
#include <math.h>
}

#include "hip/hip_runtime_api.h"

typedef struct {
  dim3 n_threads;
  dim3 n_blocks;
} KernelLaunchParams;

KernelLaunchParams get_vector_launch_parametes(CUDADevice *device,
                                               int vec_dims) {
  // Simple algorithm to find an appropriate
  //  number of blocks/threads based on device.
  int max_threads = device->max_threads_per_block;
  int n_threads = max_threads > vec_dims ? vec_dims : max_threads;
  int n_blocks = 1 + (int)ceil((vec_dims - n_threads) / n_threads);

  return {dim3(n_threads), dim3(n_blocks)};
}

KernelLaunchParams get_maitrx_launch_parametes(CUDADevice *device, int mat_rows,
                                               int mat_columns) {
  // Simple algorithm to find an appropriate
  //  number of blocks/threads based on device.
  int max_threads = device->max_threads_per_block;
  int max_threads_dim = (int)floor(sqrt(max_threads));
  dim3 n_threads(max_threads_dim > mat_rows ? mat_rows : max_threads_dim,
                 max_threads_dim > mat_columns ? mat_columns : max_threads_dim);
  dim3 n_blocks(1 + (int)ceil((mat_rows - n_threads.x) / n_threads.x),
                1 + (int)ceil((mat_columns - n_threads.y) / n_threads.y));
  return {n_threads, n_blocks};
}

extern "C" Vector *cpu_gpu_conditional_apply_vector_operator(
    void (*cpu_op)(Vector *, Vector *, Vector *),
    void (*gpu_op)(Vector *, Vector *, Vector *),
    bool (*validate)(Vector *, Vector *, Vector *), Vector *a, Vector *b,
    Vector *dst, int alloc_dims, CUDADevice *alloc_device) {
  // Allocate destination Vector if needed
  dst = maybe_alloc_vector(dst, alloc_dims, alloc_device);

  // Assert pre-conditions
  assert(validate(a, b, dst));

  // Apply operation
  if (a->device == NULL) {
    // If it's CPU, just call it directly
    cpu_op(a, b, dst);
  } else {
    KernelLaunchParams params =
        get_vector_launch_parametes(dst->device, dst->dims);

    // Launch the kernel with the cu_vectors
    gpu_op<<<params.n_blocks, params.n_threads>>>(a->cu_vector, b->cu_vector,
                                                  dst->cu_vector);
  }

  // Return dst
  return dst;
}

Vector *cpu_gpu_conditional_apply_scalar_vector_operator(
    void (*cpu_op)(double *, Vector *, Vector *),
    void (*gpu_op)(double *, Vector *, Vector *), double a, Vector *b,
    Vector *dst, int alloc_dims, CUDADevice *alloc_device) {
  // Allocate destination Matrix if needed
  dst = maybe_alloc_vector(dst, alloc_dims, alloc_device);

  // Apply operation
  if (b->device == NULL) {
    // If it's CPU, just call it directly
    cpu_op(&a, b, dst);
  } else {
    CUDADevice *device = dst->device;
    KernelLaunchParams params = get_vector_launch_parametes(device, dst->dims);

    // Allocate temporary memory for double
    double *cu_a = NULL;
    hipMalloc(&cu_a, sizeof(double));
    hipMemcpy(cu_a, &a, sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel with the cu_vectors
    gpu_op<<<params.n_blocks, params.n_threads>>>(cu_a, b->cu_vector,
                                                  dst->cu_vector);

    // Deallocate memory
    hipFree(cu_a);
  }

  // Return dst
  return dst;
}

extern "C" Matrix *cpu_gpu_conditional_apply_matrix_operator(
    void (*cpu_op)(Matrix *, Matrix *, Matrix *),
    void (*gpu_op)(Matrix *, Matrix *, Matrix *),
    bool (*validate)(Matrix *, Matrix *, Matrix *), Matrix *a, Matrix *b,
    Matrix *dst, int alloc_rows, int alloc_columns, CUDADevice *alloc_device) {
  // Allocate destination Matrix if needed
  dst = maybe_alloc_matrix(dst, alloc_rows, alloc_columns, alloc_device);

  // Assert pre-conditions
  assert(validate(a, b, dst));

  // Apply operation
  if (a->device == NULL) {
    // If it's CPU, just call it directly
    cpu_op(a, b, dst);
  } else {
    KernelLaunchParams params =
        get_maitrx_launch_parametes(dst->device, dst->rows, dst->columns);

    // Launch the kernel with the cu_matrices
    gpu_op<<<params.n_blocks, params.n_threads>>>(a->cu_matrix, b->cu_matrix,
                                                  dst->cu_matrix);
  }

  // Return dst
  return dst;
}

extern "C" Matrix *cpu_gpu_conditional_apply_scalar_matrix_operator(
    void (*cpu_op)(double *, Matrix *, Matrix *),
    void (*gpu_op)(double *, Matrix *, Matrix *), double a, Matrix *b,
    Matrix *dst, int alloc_rows, int alloc_columns, CUDADevice *alloc_device) {
  // Allocate destination Matrix if needed
  dst = maybe_alloc_matrix(dst, alloc_rows, alloc_columns, alloc_device);

  // Apply operation
  if (b->device == NULL) {
    // If it's CPU, just call it directly
    cpu_op(&a, b, dst);
  } else {
    CUDADevice *device = dst->device;
    KernelLaunchParams params =
        get_maitrx_launch_parametes(device, dst->rows, dst->columns);

    // Allocate temporary memory for double
    double *cu_a = NULL;
    hipMalloc(&cu_a, sizeof(double));
    hipMemcpy(cu_a, &a, sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel with the cu_matrices
    gpu_op<<<params.n_blocks, params.n_threads>>>(cu_a, b->cu_matrix,
                                                  dst->cu_matrix);

    // Deallocate memory
    hipFree(cu_a);
  }

  // Return dst
  return dst;
}
